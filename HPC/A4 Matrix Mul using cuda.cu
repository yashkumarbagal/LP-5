#include <iostream>
#include <hip/hip_runtime.h>

#define N 16  // Size of square matrices (N x N)

// CUDA kernel for matrix multiplication
__global__ void matrixMul(float* A, float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid size
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + 15) / 16, (N + 15) / 16);

    // Launch kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result matrix (first 4x4 part)
    std::cout << "Result matrix (partial 4x4):\n";
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << "\n";
    }

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

//nvcc matrix_mul.cu -o matrix_mul
// ./matrix_mul
