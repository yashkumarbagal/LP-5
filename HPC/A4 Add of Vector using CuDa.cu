#include <iostream>
#include <hip/hip_runtime.h>

#define N 1000000  // Size of vectors

// CUDA Kernel for vector addition
__global__ void vectorAdd(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print first 10 results
    std::cout << "First 10 elements of result:\n";
    for (int i = 0; i < 10; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

//Compile using:  nvcc -o vector_add vector_add.cu
// run it: ./vector_add
